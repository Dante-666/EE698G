#include "hip/hip_runtime.h"
#include "parallel.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void initialize(u32 *d_out, u32 value){
    u32 id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id > MAX_BINS - 1) return;
    d_out[id] = value;
}

__global__ void m_hist(u8 *d_in, u32 *d_out, u32 length) {
    u32 id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id > length - 1) return;
    __shared__ u32 bin[MAX_BINS];

    if (threadIdx.x < MAX_BINS) bin[threadIdx.x] = 0;
    
    bin[d_in[id]]++;

    if (threadIdx.x < MAX_BINS - 1) atomicAdd(&d_out[threadIdx.x], bin[threadIdx.x]); 
}

int m_histogram(u8 *h_in, u32 *h_out, u32 length) {
    /**
     * Allocate memory for the bins [0-255]
     */
    u32 *d_out;
    hipMalloc((void **) &d_out, MAX_BINS * sizeof(u32));
    /**
     * Call kernel initialize() to initialize all values to 0.
     */
    dim3 grid = dim3((int) ceil((float) MAX_BINS/MAX_THREADS));
    dim3 block = dim3(MAX_THREADS, 1, 1);
    printf("%d\n", MAX_BINS);
    printf("%d\n", MAX_THREADS);
    printf("%f\n",ceil((float) MAX_BINS/MAX_THREADS));
    initialize<<<grid, block>>>(d_out, 0);
    /**
     * Copy the host data to machine.
     */
    u8 *d_in;
    hipMalloc((void **) &d_in, length * sizeof(u8));
    gpuErrchk(hipMemcpy(d_in, h_in, length * sizeof(u8), hipMemcpyHostToDevice));
    /**
     * Call kernel m_hist() to count the individual values.
     */
    grid = dim3(ceil(length/MAX_THREADS));
    m_hist<<<grid, block>>>(d_in, d_out, length);
    hipMemcpy(h_out, d_out, MAX_BINS * sizeof(u32), hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);

    return 0;
}
